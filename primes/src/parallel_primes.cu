#include "hip/hip_runtime.h"
#include "parallel_primes.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <cctype>
#include <cmath>
#include <cstdlib>


__global__ void is_divisible(int* is_d, int number){

  unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;
  if ((i+2)*(i+2) <= number) {
    if (number%(i+2) == 0) is_d[i] = true;
    else is_d[i] = false;
  }else is_d[i] = false;
  
}


__device__ bool is_prime(unsigned long n){
  
  if (n < 2) return false;
  if (n == 2) return true;

  for (int i = 2; i*i <= n; i++){
    if (n % i == 0) return false;
  }
  
  return true;
  
}

__global__ void count_primes_kernel( int *collector, int chunk_size, int chunk_count, unsigned long long int limit ) {

  unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;

  int local_count = 0;
  if (i < chunk_count){
    for (int k = i*chunk_size; k < (i+1)*chunk_size; k++){
      if (is_prime(k)) local_count++;
    }
  }else if (i == chunk_count && i*chunk_size <= limit){
    for (int k = i*chunk_size; k <= limit; k++){
      if (is_prime(k)) local_count++;
    }
  }
  collector[i] = local_count;
    
}

__global__ void count_primes_kernel_micro( int *collector, unsigned long long int limit ) {

  unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i <= limit){
    if (is_prime(i+1)) collector[i] = 1;
  }
    
}

int parallel::count_primes(unsigned long long n){

  int chunk_size = 100;
  int chunk_count = floor(n/chunk_size);
  // printf("Chunk count %d\n", chunk_count);

  size_t size = (chunk_count+1) * sizeof(int);
  int *collector_device = NULL;
  hipMalloc((void **)&collector_device, size);

  int *collector_host = (int*) malloc(size);

  for (size_t i = 0; i < chunk_count+1; i++){
    collector_host[i] = 0;
  }
  

  // printf("Copy input data from the host memory to the CUDA device\n");
  hipMemcpy(collector_device, collector_host, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 1024;
  unsigned long long int blocksPerGrid = (chunk_count + threadsPerBlock - 1) / threadsPerBlock;
  // printf("CUDA kernel launch with %llu blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  count_primes_kernel<<<blocksPerGrid,threadsPerBlock>>>(collector_device, chunk_size, chunk_count, n);

  hipMemcpy(collector_host, collector_device, size, hipMemcpyDeviceToHost);

  hipFree(collector_device);

  int count = 0;
  for (size_t i = 0; i < chunk_count+1; i++){
    // printf("%d ",collector_host[i]);
    count += collector_host[i];
  }
  // printf("\n");

  free(collector_host);

  return count;
   
}
