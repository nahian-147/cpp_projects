#include <stdio.h>
#include <stdlib.h>
#include "genome/parallel.cuh"
#include "hip/hip_runtime.h"


unsigned long long int str_size(char* s){
  unsigned long long int n = 0;
  while(s[n] != '\0'){
    n++;
  }
  return n;
}

void random_strand(char* s, unsigned long long int n){
  const char A[5] = {'A', 'T', 'C', 'G'};
  for(unsigned long long int i=0; i<n-1; i++){
    *(s+i) = A[rand() % 4];
  }
  *(s+n-1) = '\0';
}

int main(int argc, char **argv){
  unsigned long long int numElements;
  if(argc >= 2){
    numElements = atoi(argv[1]);
  }else
    numElements = 3000000;
  
  size_t size = numElements * sizeof(char);

  char *h_A = (char *)malloc(size);

  char *h_C = (char *)malloc(size);

  random_strand(h_A, numElements);
  printf("Size of strand = %llu\n", str_size(h_A));
  for(unsigned long long int k = 0; k < 15; k++){
    printf("%c",h_A[k]);
  }
  printf("....");
  for(unsigned long long int k = numElements-15; k < numElements; k++){
    printf("%c",h_A[k]);
  }
  printf("\n");

  unsigned long long int strand_size = str_size(h_A);

  parallel::translate(h_A, h_C, numElements, size, strand_size);
  // hipDeviceSynchronize();

  for(unsigned long long int k = 0; k < 15; k++){
    printf("%c",h_C[k]);
  }
  printf("....");
  for(unsigned long long int k = numElements-15; k < numElements; k++){
    printf("%c",h_C[k]);
  }
  printf("\n");


  // char* ptr;
  // random_strand(ptr, 15);
  // parallel::rand_str(ptr, 25);

  // for(int i=0; i<15; i++){
  //   printf("%c",ptr[i]);
  // }
  // printf("\n");

  return 0;
}
