#include <iostream>
#include <map>
#include <string>
#include <cstring>
#include <cctype>
#include "cuGENOME/parallel.h"
#include <hip/hip_runtime.h>


int main( void ) {
    int c;
    int *dev_c;

    hipMalloc( (void**)&dev_c, sizeof(int) );
    parallel::add<<<1,1>>>( 1, 7, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost );

    hipFree( dev_c );
    std::cout << c << std::endl;

}