#include "hip/hip_runtime.h"
#include "parallel.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <cctype>


__global__ void parallel::translate_kernel(const char *A, char *C, unsigned long long int num_elements){
  
  unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < num_elements){
  switch (A[i]){
    case 'A':
      C[i] = 'T';
      break;

    case 'T':
      C[i] = 'A';
      break;
                   
    case 'C':
      C[i] = 'G';
      break;
    
    case 'G':
      C[i] = 'C';
      break;               
    
    default:
      C[i] = ' ';
  }
  }
}

__global__ void parallel::rand_str_kernel( char *ptr, unsigned long long int num_elements ) {

    unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num_elements){
        switch (ptr[i]){
            case 'A':
            ptr[i] = 'a';
            break;

            case 'T':
            ptr[i] = 't';
            break;
                        
            case 'C':
            ptr[i] = 'c';
            break;
            
            case 'G':
            ptr[i] = 'g';
            break;               
            
            default:
            ptr[i] = ' ';
        }
  }
    
}

__global__ void parallel::add( int a, int b, int *c ) {
    *c = a + b;
}


void parallel::translate(char *h_A, char *h_C, unsigned long long int numElements, size_t size, unsigned long long int strand_size){

  hipError_t err = hipSuccess;

  char *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  char *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 1024;
  unsigned long long int blocksPerGrid = (strand_size + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %llu blocks of %d threads\n", blocksPerGrid,
          threadsPerBlock);
  parallel::translate_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_C, strand_size);
  // hipDeviceSynchronize();

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}


char* parallel::rand_str( char *ptr, unsigned long long int num_elements ){
    hipError_t err = hipSuccess;

    char *d_ptr = NULL;
    size_t size = num_elements * sizeof(char);
    err = hipMalloc((void **)&d_ptr, size);
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_ptr, ptr, size, hipMemcpyHostToDevice);

    parallel::rand_str_kernel<<<3,5>>>(ptr, num_elements);

    hipMemcpy(ptr, d_ptr, size, hipMemcpyDeviceToHost);
    hipFree(d_ptr);
}