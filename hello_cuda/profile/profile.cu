#include <cstdio>

int main( void ) {
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount( &count );
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "--- General Information for device %d ---\n", i );
        printf( "Name:%s\n", prop.name );
        printf( "Compute capability:%d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:%d\n", prop.clockRate );
        printf( "Device copy overlap:" );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );
            printf( "Kernel execition timeout : " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );
            printf( "--- Memory Information for device %d ---\n", i );
            printf( "Total global mem:%ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:%ld\n", prop.totalConstMem );
        printf( "Max mem pitch:%ld\n", prop.memPitch );
        printf( "Texture Alignment:%ld\n", prop.textureAlignment );

        printf( "maxTexture1D:%ld\n", prop.maxTexture1D );
        printf( "maxTexture2D:%ld\n", prop.maxTexture2D );
        printf( "maxTexture3D:%ld\n", prop.maxTexture3D );

        if(prop.concurrentKernels){
            printf("Concurrent Kernel: Supported.\n");
        }else
            printf("Concurrent Kernel: Not Supported\n");

        if (prop.canMapHostMemory){
            printf("Can Map Host Memory.\n");
        }else
            printf("Can't Map Host Memory.\n");
        

        printf( "--- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:%d\n",prop.multiProcessorCount );
        printf( "Shared mem per mp:%ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:%d\n", prop.regsPerBlock );
        printf( "Threads in warp:%d\n", prop.warpSize );
        printf( "Max threads per block:%d\n",
        prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:(%d, %d, %d)\n",
        prop.maxThreadsDim[0], prop.maxThreadsDim[1],
        prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:(%d, %d, %d)\n",
        prop.maxGridSize[0], prop.maxGridSize[1],
        prop.maxGridSize[2] );
        printf( "\n" );
    }
}